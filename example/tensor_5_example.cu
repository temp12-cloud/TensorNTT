#include "common.cuh"
#include "modular.cuh"
#include "parameters.cuh"
#include "tensor_ntt.cuh"
#include <cstdint>
#include <cstring>
#include <iostream>
#include <sys/types.h>
#include <vector>


int main(int argc, char* argv[])
{
    CudaDevice(); 
    int log_nttSize = argc > 1 ? atoi(argv[1]) : 10;
    int batchSize = argc > 2 ? atoi(argv[2]) : 1;
    TensorNTTParameters parameters(log_nttSize, 5);
    std::cout << "batchSize: " << batchSize << std::endl;
    std::cout << "log_nttSize: " << parameters.log_nttSize << std::endl;
    std::cout << "nttSize: " << parameters.nttSize << std::endl;
    std::cout << "majorSize: " << parameters.majorSize << std::endl;
    std::cout << "minorSize: " << parameters.minorSize << std::endl;
    std::cout << "modulus: " << parameters.modulus.value << std::endl;

    std::vector<uint64_t> h_input(batchSize * parameters.nttSize, 1);
    std::vector<uint64_t> h_input1(batchSize * parameters.nttSize, 0);
    std::vector<uint64_t> h_output(batchSize * parameters.nttSize, 0);
    uint64_t* d_input;
    uint64_t* d_output;
    uint8_t* d_majorMatrix;
    uint8_t* d_minorMatrix;
    uint64_t* d_factorTable;

    CUDA_CHECK(hipMalloc((void**)&d_input, h_input.size() * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc((void**)&d_output, h_output.size() * sizeof(uint64_t)));
    CUDA_CHECK(hipMalloc((void**)&d_majorMatrix, parameters.majorMatrix.size() * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&d_minorMatrix, parameters.minorMatrix.size() * sizeof(uint8_t)));
    CUDA_CHECK(hipMalloc((void**)&d_factorTable, parameters.factorTable.size() * sizeof(uint64_t)));

    CUDA_CHECK(hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(uint64_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_majorMatrix, parameters.majorMatrix.data(), parameters.majorMatrix.size() * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_minorMatrix, parameters.minorMatrix.data(), parameters.minorMatrix.size() * sizeof(uint8_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_factorTable, parameters.factorTable.data(), parameters.factorTable.size() * sizeof(uint64_t), hipMemcpyHostToDevice));

    GPU_TensorNTT_5(d_input, d_output, d_majorMatrix, d_minorMatrix, d_factorTable, parameters.log_nttSize, parameters.modulus, batchSize, parameters.nttSize);

    CUDA_CHECK(hipMemcpy(h_output.data(), d_output, h_output.size() * sizeof(uint64_t), hipMemcpyDeviceToHost));
    for (int i = 0; i < batchSize; i++)
    {
        for (int j = 0; j < 32*4; j++)
        {
            for (int k = 0; k < 32; k++)
            {
                std::cout << h_output[i * parameters.nttSize + j * 32 + k] << " ";
            }
            if (j % 32 == 31)
            {
                std::cout << std::endl;
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    return 0;
}
