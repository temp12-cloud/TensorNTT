// 本文件用于测试merge_ntt.cu中的准确性，从而为后续测试提供正确性参考。
#include "merge_ntt.cuh"
#include "parameters.cuh"
#include <cstdint>
#include <iostream>

int main(int argc, char* argv[])
{

    int log_nttSize = argc > 1 ? atoi(argv[1]) : 12;
    
    
    NTTParameters parameters(log_nttSize);

    std::vector<uint64_t> input(parameters.nttSize, 1);
    std::vector<uint64_t> output_GPU(parameters.nttSize, 0);
    std::vector<uint64_t> output_CPU(parameters.nttSize, 0);
    std::cout << "log_nttSize: " << parameters.log_nttSize << std::endl;
    std::cout << "nttSize: " << parameters.nttSize << std::endl;
    std::cout << "rootSize: " << parameters.rootSize << std::endl;

    


    uint64_t *d_input, *d_output, *d_unityRootReverseTable, *d_inverseRootUnityReverseTable;
    Modulus* d_modulus;

    hipMalloc(&d_input, parameters.nttSize * sizeof(uint64_t));
    hipMalloc(&d_output, parameters.nttSize * sizeof(uint64_t));
    hipMalloc(&d_unityRootReverseTable, parameters.rootSize * sizeof(uint64_t));
    hipMalloc(&d_inverseRootUnityReverseTable, parameters.rootSize * sizeof(uint64_t));
    hipMalloc(&d_modulus, sizeof(Modulus));

    hipMemcpy(d_input, input.data(), parameters.nttSize * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_unityRootReverseTable, parameters.unityRootReverseTable.data(), parameters.rootSize * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_inverseRootUnityReverseTable, parameters.inverseUnityRootReverseTable.data(), parameters.rootSize * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipStream_t stream0;
    hipStreamCreate(&stream0);

    GPU_NTT(d_input, d_output, d_unityRootReverseTable, log_nttSize, parameters.modulus, 1, stream0);

    hipMemcpy(output_GPU.data(), d_output, parameters.nttSize * sizeof(uint64_t), hipMemcpyDeviceToHost);

    for (int i = 0; i < parameters.nttSize; i++)
    {
        std::cout << output_GPU[i] << " ";
    
    }
    return 0;
}
